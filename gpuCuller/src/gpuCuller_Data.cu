#include <gpuCuller_internal.h>

#include <thrust/device_ptr.h>

//-------- Data References --------
thrust::device_ptr<aabb_t> d_AABB;
thrust::device_ptr<bvhnode_t> d_BVHNODE;
thrust::device_ptr<lbvhsplit_t> d_SPLITSLIST;
thrust::device_ptr<hnode_t> d_HIERARCHY;
unsigned int universeElementCount;
unsigned int bvhDepth;
aabb_t universeAABB;
//---------------------------------