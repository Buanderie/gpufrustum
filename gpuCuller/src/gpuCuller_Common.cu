#include <cutil_inline.h>
#include <gpuCuller.h>

void __stdcall gculInitialize( int argc, char** argv )
{
	// Initializes CUDA device
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );
}