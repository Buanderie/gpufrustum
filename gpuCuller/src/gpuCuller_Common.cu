#include <windows.h>
#include <cutil_inline.h>
#include <gpuCuller.h>
#include <gpuCuller_internal.h>
#include <thrust/device_vector.h>
#include <iostream>

using namespace std;

void __stdcall gculInitialize( int argc, char** argv )
{
	printf("Initializing CUDA...\n");
	// Initializes CUDA device
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );
	printf("CUDA Initialized, using device #%i\n", cutGetMaxGflopsDeviceId());
}

void __stdcall gculLoadAABB( unsigned int N, const void* ptr )
{
	//Load AABB data onto Device
	aabb_t * aabb_raw_ptr;
    hipMalloc((void **) &aabb_raw_ptr, N * sizeof(aabb_t));
	hipMemcpy(aabb_raw_ptr, ptr, sizeof(aabb_t)*N, hipMemcpyHostToDevice);
	d_AABB = thrust::device_ptr<aabb_t>(aabb_raw_ptr);

	//Prepare memory for BVH Nodes
	bvhnode_t * bvhnode_raw_ptr; 
	hipMalloc((void **) &bvhnode_raw_ptr, N * sizeof(bvhnode_t));
	d_BVHNODE = thrust::device_ptr<bvhnode_t>(bvhnode_raw_ptr);
}

void __stdcall gculBuildLBVH()
{
	//First step: Assign Morton Codes to BVH Nodes

}